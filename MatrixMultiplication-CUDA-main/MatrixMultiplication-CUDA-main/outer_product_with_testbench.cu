#include "hip/hip_runtime.h"

// Import required header files
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Kernel function for outer-product-based matrix multiplication
__global__ void outerProductKernel(float *A, float *B, float *C, int N) {
    // Calculate row and column indices for the thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Declare a variable to accumulate the sum
    float sum = 0;
    
    // Check if the thread's indices are within the matrix dimensions
    if(row < N && col < N) {
        // Compute the dot product for the i-th row of A and the j-th column of B
        for(int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        // Store the sum in the corresponding element of matrix C
        C[row * N + col] = sum;
    }
}

// Function to test the outer product implementation
void testOuterProduct(float *A, float *B, float *C, int N) {
    // Declare device pointers for matrices A, B, and C
    float *d_A, *d_B, *d_C;

    // Allocate memory on the GPU for matrices A, B, and C
    hipMalloc((void **)&d_A, N * N * sizeof(float));
    hipMalloc((void **)&d_B, N * N * sizeof(float));
    hipMalloc((void **)&d_C, N * N * sizeof(float));

    // Copy the data for matrices A and B from host to device
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Define the number of threads per block and the number of blocks per grid
    dim3 threadsPerBlock(2, 2);
    dim3 blocksPerGrid(N / threadsPerBlock.x, N / threadsPerBlock.y);

    // Launch the kernel
    outerProductKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy the resultant matrix C from device to host
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free the allocated device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Main function
int main() {
    // Define the dimension of the matrices
    int N = 4;
    // Initialize matrices A and B
    float A[N * N] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    float B[N * N] = {16, 15, 14, 13, 12, 11, 10, 9, 8, 7, 6, 5, 4, 3, 2, 1};
    // Declare the result matrix C
    float C[N * N];

    // Run the test function
    testOuterProduct(A, B, C, N);

    // Print the resultant matrix C
    printf("Matrix C: \n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", C[i * N + j]);
        }
        printf("\n");
    }

    // Exit the program
    return 0;
}
