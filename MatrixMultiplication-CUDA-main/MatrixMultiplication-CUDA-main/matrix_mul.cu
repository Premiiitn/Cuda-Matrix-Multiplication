
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void innerProductKernel(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) {
        float sum = 0;
        for(int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    int N = 16;
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    A = (float *)malloc(N * N * sizeof(float));
    B = (float *)malloc(N * N * sizeof(float));
    C = (float *)malloc(N * N * sizeof(float));

    // Initialize matrices A and B here

    hipMalloc((void **)&d_A, N * N * sizeof(float));
    hipMalloc((void **)&d_B, N * N * sizeof(float));
    hipMalloc((void **)&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(N / threadsPerBlock.x, N / threadsPerBlock.y);
    innerProductKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C);

    return 0;
}
